#include "macro.h"

/**
 * The function finds a nnf from a to b, it's not guaranteed that all pixels in b has a match in a.
 * a, b, b_prime are all known
 *
 * @param a
 * @param b
 * @param a_prime
 * @param b_prime
 * @param width
 * @param height
 * @param patch_size
 * @param u
 * @param num_iterations
 * @param nnf_from_a
 */
void patchMatch(float* a, float* b, float* a_prime, float* b_prime, int width, int height, int patch_size, int u,
                int num_iterations, int* nnf_from_a)
{
    // Allocate device memory
    float* dev_a;
    float* dev_b;
    float* dev_a_prime;
    float* dev_b_prime;

    hipMalloc(&dev_a, width * height * sizeof(float));
    hipMalloc(&dev_b, width * height * sizeof(float));
    hipMalloc(&dev_a_prime, width * height * sizeof(float));
    hipMalloc(&dev_b_prime, width * height * sizeof(float));

    hipMemcpy(dev_a, a, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, width * height * sizeof(float), hipMemcpyHostToDevice);

    // An initial value of A' is the same as A, meaning no style transfer
    hipMemcpy(dev_a_prime, a_prime, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b_prime, b_prime, width * height * sizeof(float), hipMemcpyHostToDevice);

    int* dev_forward_nnf;
    hipMalloc(&dev_forward_nnf, 2 * width * height * sizeof(int));

    //initialize_nnf << <dim3((width - 1) / BLOCK_SIZE + 1, (height - 1) / BLOCK_SIZE + 1), dim3(BLOCK_SIZE, BLOCK_SIZE) >> > (
    //    dev_forward_nnf, width, height, patch_size, 5206);
    //    int* dev_backward_nnf;
    //    hipMalloc(&dev_backward_nnf, 2 * width * height * sizeof(int));
    //
    //    initialize_nnf<<<dim3((width - 1) / BLOCK_SIZE + 1, (height - 1) / BLOCK_SIZE + 1), dim3(BLOCK_SIZE, BLOCK_SIZE)>>>(
    //            dev_backward_nnf, width, height, 5206);

    float* dev_distances;
    hipMalloc(&dev_distances, width * height * sizeof(float));

    // Main loop
    for (int j = 0; j < num_iterations; j++) {
        initialize_nnf << <dim3((height - 1) / BLOCK_SIZE + 1, (width - 1) / BLOCK_SIZE + 1), dim3(BLOCK_SIZE, BLOCK_SIZE) >> > (
                dev_forward_nnf, width, height, patch_size, 5206);
        for (int i = 0; i < 6; i++) {
            apply_nnf << <dim3((height - 1) / BLOCK_SIZE + 1, (width - 1) / BLOCK_SIZE + 1), dim3(BLOCK_SIZE, BLOCK_SIZE) >> > (
                    dev_a_prime, dev_b_prime, width, height, patch_size, u, dev_forward_nnf);
            hipDeviceSynchronize();
            compute_patch_distances << <dim3((height - 1) / BLOCK_SIZE + 1, (width - 1) / BLOCK_SIZE + 1), dim3(BLOCK_SIZE, BLOCK_SIZE) >> > (
                    dev_a, dev_b, dev_a_prime, dev_b_prime, width, height,
                            patch_size, u, dev_forward_nnf, dev_distances);
            hipDeviceSynchronize();
            propagate << <dim3((height - 1) / BLOCK_SIZE + 1, (width - 1) / BLOCK_SIZE + 1), dim3(BLOCK_SIZE, BLOCK_SIZE) >> > (
                    dev_a, dev_b, width, height, dev_distances, dev_forward_nnf, patch_size, i % 2 == 1);
            hipDeviceSynchronize();
            //        if (i % 2 == 0) {
            //            compute_patch_distances<<<dim3((width - 1) / BLOCK_SIZE + 1, (height - 1) / BLOCK_SIZE + 1),dim3(BLOCK_SIZE, BLOCK_SIZE)>>>(
            //                    dev_a, dev_b, dev_a_prime, dev_b_prime, width, height,
            //                    patch_size, u, dev_forward_nnf, dev_distances);
            //
            //            propagate<<<dim3((width - 1) / BLOCK_SIZE + 1, (height - 1) / BLOCK_SIZE + 1),dim3(BLOCK_SIZE, BLOCK_SIZE)>>>(
            //                    dev_a, dev_b, width, height, dev_distances, dev_forward_nnf, patch_size, false);
            //        }
            //        else {
            //            compute_patch_distances<<<dim3((width - 1) / BLOCK_SIZE + 1, (height - 1) / BLOCK_SIZE + 1),dim3(BLOCK_SIZE, BLOCK_SIZE)>>>(
            //                    dev_b, dev_a, dev_a_prime, dev_b_prime, width, height,
            //                    patch_size, u, dev_forward_nnf, dev_distances);
            //            propagate<<<dim3((width - 1) / BLOCK_SIZE + 1, (height - 1) / BLOCK_SIZE + 1),dim3(BLOCK_SIZE, BLOCK_SIZE)>>>(
            //                    dev_b, dev_a, width, height, dev_distances, dev_forward_nnf, patch_size, true);
            //        }
        }
    }


    // Copy result back to host
    hipMemcpy(nnf_from_a, dev_forward_nnf, 2 * width * height * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(a_prime, dev_a_prime, width * height * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_a_prime);
    hipFree(dev_b_prime);
    hipFree(dev_forward_nnf);
    //    hipFree(dev_backward_nnf);
    hipFree(dev_distances);

}
